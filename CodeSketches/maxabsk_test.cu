#define BLOCK_SIZE 1024

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// the reduction operator
// a and b are K-element arrays sorted in descending magnitude i.e. [5 -4 3 1 0 0]
// buf is a K-element array used as buffer to store results before copying back to a
// index is copied along in the same order as its corresponding element.
/*
 Example Input:
 a : [5 -4 3 1 0 0]
 b : [-9 2 -2 1 0 0]
 buf : don't care
 Example Output:
 a=buf : [-9 5 -4 3 2 -2]
 b : [-9 2 -2 1 0 0]
*/
// note that the data type needs to be integral!
template <typename T>
__device__ void merge_maxabs(T* a, T *b,int* idxA, int* idxB, T* buf, int* idxbuf, const int N, const int K)
{
	int ia = 0;
	int ib = 0;
	for(int i=0;i<K;i++)
	{
		T aval = a[ia];		 T bval = b[ib];
		int aidx = idxA[ia];	 int bidx = idxB[ib];
		T amag = fabs(aval); 	 T bmag = fabs(bval);
		int isequal = amag==bmag;
		int incb = isequal ? (bidx<aidx) : (bmag>amag);
		buf[i] = incb ? bval : aval;
		idxbuf[i] = incb ? bidx : aidx;
		ia += !incb;
		ib += incb;
	}
	for(int i=0;i<K;i++)
	{
		a[i] = buf[i];
		idxA[i] = idxbuf[i];
	}
}
// thresholding on BLOCK_SIZE chunks of data in a sample
// can be used recursively on output until K max magnitudes are left
// Do we need to preserve information about the order of elements?
// input[N]
// output[num_blocks x K]
// output is K max magnitudes of each block, in descending order
template <typename T>
__global__ void max_abs_k(T *input, T *output, int* idxinput, int init_index, const int N, const int K) {  
  extern __shared__ T data[];
  int tx = threadIdx.x, bx = blockIdx.x;
  int offset = tx*K;
  int* idxdata = (int*) &data[BLOCK_SIZE*K];
  T* buf = (T*) &idxdata[BLOCK_SIZE*K];
  int* idxbuf = (int*) &buf[BLOCK_SIZE*K];
  unsigned int i = bx*BLOCK_SIZE + tx;
  data[offset] = i<N ? input[i] : 0;       
  idxdata[offset] = i<N ? ( init_index ? i : idxinput[i] ): N;       
  for(int j=1;j<K;j++)
  {
	data[offset+j] = 0;	    	
	idxdata[offset+j] = N;
  }		
  __syncthreads();  
  for(unsigned int stride=(BLOCK_SIZE>>1);stride>0;stride>>=1)
  {	  
	  if(tx<stride){
		//perform min absolute K reduction operator
		int stride_offset = offset+stride*K;
		merge_maxabs(data+offset,data+stride_offset,idxdata+offset, idxdata+stride_offset, buf+offset, idxbuf+offset,N,K);  	 
	  }
	  __syncthreads();
  }  
  if(tx<K)
  {
	output[bx*K+tx] = data[tx];
	idxinput[bx*K+tx] = idxdata[tx];
  }  
}

//
// kernel call wrapper. Since this is a sketch it will only handle one sample at a time.
// One sample is probably already large enough for a GPU, so maybe each sample (or a fixed limited number of them) should go to a different GPU on different machines?
// input : N
// output : num_blocks x K
template <typename T>
__host__ void do_max_abs_k(T *initial_input, T *output,int* idxinput, const int N, const int K)
{
	int n = N;
	T* input = initial_input;		
	int shared_size_bytes = (sizeof(T)+sizeof(int))*2*BLOCK_SIZE*K; // 4 parts: data(T), index data(int), data buffer(T), index buffer (int)
	int num_blocks = (int) ceil(n*1.0/BLOCK_SIZE);		
	int init_index = 1; //use direct index for the first round
	do
	{		
		max_abs_k<<<num_blocks,BLOCK_SIZE,shared_size_bytes>>>(input,output,idxinput,init_index,n,K);
		init_index = 0;
		n = K*num_blocks;
		num_blocks = (int) ceil(n*1.0/BLOCK_SIZE);		
		input = output;
	}while(n!=K);
	// now output[0..K-1] contains the max k elements 
	// now idxinput[0..K-1] storing corresponding index (index order preserved)
}

int deviceQuery()
{
  int deviceCount;

  hipGetDeviceCount(&deviceCount);

  printf("start - Getting GPU Data.\n"); //@@ start a timer

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        printf("No CUDA GPU has been detected\n");
        return -1;
      } else if (deviceCount == 1) {
        printf("There is 1 device supporting CUDA\n");
      } else {
        printf("There are %d devices supporting CUDA\n", deviceCount);
      }
    }
    printf("Device %d: %s\n", dev, deviceProp.name);
    printf("Computational Capabilities: %d.%d\n", deviceProp.major, deviceProp.minor);    
    printf(" Maximum global memory size: %lu\n", deviceProp.totalGlobalMem);
    printf(" Maximum constant memory size: %lu\n", deviceProp.totalConstMem);
    printf(" Maximum shared memory size per block: %d\n", (int)deviceProp.sharedMemPerBlock);
    printf(" Maximum block dimensions: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf(" Maximum grid dimensions: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf(" Warp size: %d\n", deviceProp.warpSize);
  }

  printf("stop - Getting GPU Data.\n"); //@@ stop the timer
  return 0;
}
template<typename K, typename V>
__host__ void mergekv(K* keys, V* values, int start, int mid, int end)
{	
	int size = end - start;
	int ia = start;
	int ib = mid;
	int i = 0;
	K* ktmp = (K*)malloc(sizeof(K)*size);
	V* vtmp = (V*)malloc(sizeof(V)*size);
	while(ia < mid && ib < end)
	{
		int inca = keys[ia] < keys[ib];
		ktmp[i] = inca ? keys[ia] : keys[ib];
		vtmp[i] = inca ? values[ia]: values[ib];
		ia += inca;
		ib += !inca;
		i++;
	}
	while(ia < mid) 
	{
		ktmp[i] = keys[ia];
		vtmp[i] = values[ia];
		ia++;
		i++;
	}
	while(ib < end)
	{
		ktmp[i] = keys[ib];
		vtmp[i] = values[ib];
		ib++;
		i++;
	}
	for(i=0;i<size;i++)
	{
		keys[start+i] = ktmp[i];
		values[start+i] = vtmp[i];
	}
	free(ktmp);
	free(vtmp);
}
template<typename K, typename V>
__host__ void sortkv(K* keys, V* values,int start, int end)
{
	int size = end-start;
	if(size<2) return;
	if(size==2) 
	{
		if(keys[start+1] < keys[start])
		{
			K tmpk = keys[start];
			keys[start] = keys[start+1];
			keys[start+1] = tmpk;
			V tmpv = values[start];
			values[start] = values[start+1];
			values[start+1] = tmpv;
		}
	} else {
		int mid = (start+end)/2;
		sortkv(keys,values,start, mid);
		sortkv(keys,values,mid, end);
		mergekv(keys,values,start,mid,end);
	}
}

/*
* Performs an in-place sort, sorted in ascending order
*/
template<typename K, typename V>
__host__ void do_sortkv(K* keys, V* values, int N)
{
	sortkv(keys,values,0,N);	
}
// TODO take this entry point away and implement entry point for matlab mex
int main() 
{
	int input_size;
	int k ;
	//get data attributes
	std::cin >> input_size;
	std::cin >> k;
	if(k<0 || input_size < 0 || k > input_size)
	{
		printf("error: input_size and k must be positive and k > input_size\n");
		return -1;
	}
	int num_blocks = (int) ceil(input_size*1.0/BLOCK_SIZE);
	int output_size = k*num_blocks;
	float* hInput = (float*) malloc(sizeof(float)*input_size);
	float* hOutput = (float*) malloc(sizeof(float)*k);
	int* hIdx = (int*) malloc(sizeof(int)*k);
	
	//populate input
	for(int i=0;i<input_size;i++) std::cin >> hInput[i];

	float* dInput;
	int* dIdxInput;
	float* dOutput;	
	hipMalloc(&dInput, sizeof(float)*input_size);
	hipMalloc(&dIdxInput, sizeof(int)*input_size);
	hipMalloc(&dOutput, sizeof(float)*output_size);	
	hipMemcpy(dInput,hInput,sizeof(float)*input_size,hipMemcpyHostToDevice);
	do_max_abs_k(dInput,dOutput,dIdxInput,input_size,k);

	hipDeviceSynchronize();
	hipMemcpy(hOutput,dOutput,sizeof(float)*k,hipMemcpyDeviceToHost);
	hipMemcpy(hIdx,dIdxInput,sizeof(int)*k,hipMemcpyDeviceToHost);
	// TODO : sort by index not really needed. The results are fine as long as they keep track of the indexes	
	do_sortkv(hIdx,hOutput,k);
	for(int i=0;i<k;i++)
	{
		printf("%d %f\n",hIdx[i],hOutput[i]);
	}
	free(hInput);
	free(hOutput);
	hipFree(dInput);
	hipFree(dIdxInput);
	hipFree(dOutput);
	return 0;
}
