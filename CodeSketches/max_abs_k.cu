#define BLOCK_SIZE 1024

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// the reduction operator
// a and b are K-element arrays sorted in descending magnitude i.e. [5 -4 3 1 0 0]
// buf is a K-element array used as buffer to store results before copying back to a
/*
 Example Input:
 a : [5 -4 3 1 0 0]
 b : [-9 2 -2 1 0 0]
 buf : don't care
 Example Output:
 a=buf : [-9 5 -4 3 2 -2]
 b : [-9 2 -2 1 0 0]
*/
__device__ void merge_maxabs(float* a, float *b,float* idxA, float* idxB, float* buf, float* idxbuf, const int N, const int K)
{
	int ia = 0;
	int ib = 0;
	for(int i=0;i<K;i++)
	{
		float aval = a[ia];		 float bval = b[ib];
		float aidx = idxA[ia];	 float bidx = idxB[ib];
		float amag = fabs(aval); float bmag = fabs(bval);
		float maxmag = fmax(amag,bmag);
		int iszeros = aval==0 && bval==0;		
		int incb = iszeros ?  aidx==N : (maxmag==bmag) && (amag!=bmag);
		
		buf[i] = incb ? bval : aval;				
		idxbuf[i] = incb ? bidx : aidx;
		ia += !incb;
		ib += incb;
	}
	for(int i=0;i<K;i++)
	{
		a[i] = buf[i];
		idxA[i] = idxbuf[i];
	}
}
// thresholding on BLOCK_SIZE chunks of data in a sample
// can be used recursively on output until K max magnitudes are left
// Do we need to preserve information about the order of elements?
// input[N]
// output[num_blocks x K]
// output is K max magnitudes of each block, in descending order
__global__ void max_abs_k(float *input, float *output, int* idxinput, int init_index, const int N, int K) {
  //__shared__ float p[BLOCK_SIZE+1][K]; /*!! kernel code cannot use variable here in array declaration, need to pass through kernel call!!*/
  extern __shared__ float p[];
  int tx = threadIdx.x, bx = blockIdx.x;
  float* data = p+ tx*K;
  float* idxdata = p+(BLOCK_SIZE+tx)*K;
  float* buf = p+(BLOCK_SIZE*2+tx)*K;
  float* idxbuf = p+(BLOCK_SIZE*3+tx)*K;
  unsigned int i = bx*BLOCK_SIZE + tx;
  data[0] = i<N ? input[i] : 0;       
  idxdata[0] = i<N ? ( init_index ? (idxinput[i]=i) : idxinput[i] ): N;       
  for(int j=1;j<K;j++)
  {
	data[j] = 0;	    	
	idxdata[j] = N;
  }		
  __syncthreads();  
  for(unsigned int stride=(BLOCK_SIZE>>1);stride>0;stride>>=1)
  {	  
	  if(tx<stride){
		//perform min absolute k
		merge_maxabs(data,data+stride*K,idxdata, idxdata+stride*K, buf, idxbuf,N,K);  	 
	  }
	  __syncthreads();
  }  
  if(tx<K)
  {
	  int count = 0;
	  float my_val = p[tx];
	  int my_idx = p[BLOCK_SIZE*K+tx];
	  for(int j=0;j<K;j++)
	  {
		  count += (my_idx > p[BLOCK_SIZE*K+j]);
	  }
	output[bx*K+count] = my_val;
	idxinput[bx*K+count] = my_idx;
  }  
}

// kernel call wrapper. Since this is a sketch it will only handle one sample at a time.
// One sample is probably already large enough for a GPU, so maybe each sample (or a fixed limited number of them) should go to a different GPU on different machines?
// input : N
// output : num_blocks x K
__host__ void do_max_abs_k(float *initial_input, float *output,int* idxinput, const int N, const int K)
{
	int n = N;
	float* input = initial_input;	
	int shared_size_bytes = sizeof(float)*BLOCK_SIZE*K*4; // three parts: data, index data, data buffer, index buffer
	int num_blocks = ceil(n*1.0/BLOCK_SIZE);		
	int init_index = 1;
	int round = 0;
	do
	{		
		printf("doing round %d: size=%d (%d blocks)\n", round, n, num_blocks);
		max_abs_k<<<num_blocks,BLOCK_SIZE,shared_size_bytes>>>(input,output,idxinput,init_index,n,K);
		init_index = 0;
		n = K*num_blocks;
		num_blocks = ceil(n*1.0/BLOCK_SIZE);		
		input = output;
	}while(n!=K);
	// now output[0..K-1] contains the max k elements 
	// now idxinput[0..K-1] storing corresponding index (index order preserved)
}

__global__ void twice(float* a, float* b, int n)
{
	int tx = threadIdx.x, bx = blockIdx.x;
	int idx = bx*BLOCK_SIZE + tx;
	if(idx < n) b[idx] = 2*a[idx];
}


int deviceQuery()
{
  int deviceCount;

  hipGetDeviceCount(&deviceCount);

  printf("start - Getting GPU Data.\n"); //@@ start a timer

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        printf("No CUDA GPU has been detected\n");
        return -1;
      } else if (deviceCount == 1) {
        printf("There is 1 device supporting CUDA\n");
      } else {
        printf("There are %d devices supporting CUDA\n", deviceCount);
      }
    }
    printf("Device %d name: %s\n", dev, deviceProp.name);
    printf("Computational Capabilities: %d.%d\n", deviceProp.major, deviceProp.minor);    
    printf(" Maximum global memory size: %d\n", deviceProp.totalGlobalMem);
    printf(" Maximum constant memory size: %d\n", deviceProp.totalConstMem);
    printf(" Maximum shared memory size per block: %d\n", deviceProp.sharedMemPerBlock);
    printf(" Maximum block dimensions: %dx%dx%d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf(" Maximum grid dimensions: %dx%dx%d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf(" Warp size: %d\n", deviceProp.warpSize);
  }

  printf("stop - Getting GPU Data.\n"); //@@ stop the timer
  return 0;
}

int main(int argc, char **argv) 
{
	if(deviceQuery())
	{
		return 0;
	}
	int input_size = 10;
	int k = 5;
	int num_blocks = floor(input_size*1.0/BLOCK_SIZE);
	int output_size = k*num_blocks;
	float* hInput = (float*) malloc(sizeof(float)*input_size);
	float* hOutput = (float*) malloc(sizeof(float)*k);
	printf("initializing input\n");
	for(int i=0;i<input_size;i++)
	{
		hInput[i] = ( i&1 ? 1 : -1 )*(0.123*i);
		printf("%f\n",hInput[i]);
	}
	printf("done\n");
	float* dInput;
	int* dIdxInput;
	float* dOutput;

	float *A,*B;
	hipMalloc(&A, sizeof(float)*input_size);
	hipMalloc(&B, sizeof(float)*input_size);
	hipMalloc(&dInput, sizeof(float)*input_size);
	hipMalloc(&dIdxInput, sizeof(int)*input_size);
	hipMalloc(&dOutput, sizeof(float)*output_size);
	hipMemcpy(A,hInput,sizeof(float)*input_size,hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	hipMemcpy(dInput,hInput,sizeof(float)*input_size,hipMemcpyHostToDevice);
	do_max_abs_k(dInput,dOutput,dIdxInput,input_size,k);
	hipDeviceSynchronize();
	hipMemcpy(hOutput,dOutput, sizeof(float)*k, hipMemcpyDeviceToHost);
	twice<<<num_blocks,BLOCK_SIZE>>>(A,B,input_size);
	hipDeviceSynchronize();
	hipMemcpy(hInput,B, sizeof(float)*input_size, hipMemcpyDeviceToHost);
	for(int i=0;i<k;i++)
	{
		printf("%f\n", hOutput[i]);
	}
	printf("=======\n");
	for(int i=0;i<input_size;i++)
	{
		printf("%f\n", hInput[i]);
	}
	free(hInput);
	free(hOutput);
	hipFree(A);
	hipFree(B);
	hipFree(dInput);
	hipFree(dIdxInput);
	hipFree(dOutput);
	return 0;
}
